#include <iostream>
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <sstream>
#include <fstream>
#include <vector>
#include <algorithm>

#define SIZE 5 // Matrix size
#define INPUTSIZE 2306451
#define BLOCKSIZE 1125 // 1125 * 12 = 144000
#define THREADPERBLOCK 128

inline hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess)
  {
    printf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
    if (result != cudaSuccess)
    {
      exit(1);
    }
  }
#endif
  return result;
}

__device__ bool constraintCheck(const int *matrix, const int *constraints, int *constraint_size, int *CSI, int *RC, int *index)
{

  for (int i = 0; i < *constraint_size; i++)
  {

    if ((SIZE * (constraints[(*CSI) + 4 * i] - 1) + constraints[(*CSI) + 4 * i + 1] - 1 == *RC) ||
        (SIZE * (constraints[(*CSI) + 4 * i + 2] - 1) + constraints[(*CSI) + 4 * i + 3] - 1 == *RC))
    {

      if (matrix[25 * (*index) + SIZE * (constraints[(*CSI) + 4 * i] - 1) + constraints[(*CSI) + 4 * i + 1] - 1] != -1 &&
          matrix[25 * (*index) + SIZE * (constraints[(*CSI) + 4 * i + 2] - 1) + constraints[(*CSI) + 4 * i + 3] - 1] != -1 &&
          matrix[25 * (*index) + SIZE * (constraints[(*CSI) + 4 * i] - 1) + constraints[(*CSI) + 4 * i + 1] - 1] <
              matrix[25 * (*index) + SIZE * (constraints[(*CSI) + 4 * i + 2] - 1) + constraints[(*CSI) + 4 * i + 3] - 1])
      {
        return false;
      }

      if (matrix[25 * (*index) + SIZE * (constraints[(*CSI) + 4 * i] - 1) + constraints[(*CSI) + 4 * i + 1] - 1] == 1 ||
          matrix[25 * (*index) + SIZE * (constraints[(*CSI) + 4 * i + 2] - 1) + constraints[(*CSI) + 4 * i + 3] - 1] == SIZE)
      {
        return false;
      }
    }
  }
  return true;
}

__device__ bool checkRowColumn(const int *matrix, int *id, int *value, int *index)
{
  for (int row = -(*id % SIZE); row < SIZE - (*id % SIZE); row++)
  {
    if (row != 0 && *value == matrix[25 * (*index) + *id + row])
    {
      return true;
    }
  }

  for (int col = -(*id / SIZE); col < SIZE - (*id / SIZE); col++)
  {
    if (col != 0 && *value == matrix[25 * (*index) + *id + SIZE * col])
    {
      return true;
    }
  }

  return false;
}

__device__ void iterative(int *matrix, int *constraints, int *constraint_size, int *constraintStartIndex, int *index)
{

  int RC = 0;
  bool keepIteration = true;
  int value;

  int valuesArray[SIZE * SIZE];
  for (int i = 0; i < SIZE * SIZE; i++)
  {
    valuesArray[i] = 0;
    if (matrix[25 * (*index) + i] == -1)
    {
      valuesArray[i] = 1;
    }
  }

  while (RC < SIZE * SIZE)
  {
    if (valuesArray[RC] != 0)
    {
      while (valuesArray[RC] <= SIZE && keepIteration)
      {
        value = valuesArray[RC];
        valuesArray[RC]++;
        if (!checkRowColumn(matrix, &RC, &value, index))
        {
          matrix[25 * (*index) + RC] = value;
          if (constraintCheck(matrix, constraints, constraint_size, constraintStartIndex, &RC, index))
          {
            keepIteration = false;
          }
        }
      }

      if (!keepIteration)
      {
        RC++;
        keepIteration = true;
      }
      else
      {
        for (int i = RC; i < SIZE * SIZE; i++)
        {
          if (valuesArray[i] != 0)
          {
            valuesArray[i] = 1;
          }
        }
        matrix[25 * (*index) + RC] = -1;
        RC--;
        while (valuesArray[RC] == 0)
        {
          RC--;
        }
      }
    }
    else
    {
      RC++;
    }
  }
}

__global__ void solver(int *matrix_in, int *constraints_in, long int *constraintSums_in, int *inputGridNumber)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < *inputGridNumber)
  {
    int sizeOfConstraint = constraintSums_in[index + 1] - constraintSums_in[index];
    int constraintStartIndex = constraintSums_in[index] * 4;

    iterative(matrix_in, constraints_in, &sizeOfConstraint, &constraintStartIndex, &index);
  }
}

int isSolved(int *&matrix, int *constraints, int ConstraintSize)
{

  for (int i = 0; i < ConstraintSize; i++)
  {
    if (matrix[SIZE * (constraints[4 * i] - 1) + constraints[4 * i + 1] - 1] < matrix[SIZE * (constraints[4 * i + 2] - 1) + constraints[4 * i + 3] - 1])
      return 0;
  }

  std::vector<int> rows;
  std::vector<int> cols;
  for (int rc = 0; rc < SIZE; rc++)
  {
    for (int s = 0; s < SIZE; s++)
    {
      rows.push_back(matrix[SIZE * rc + s]);
      cols.push_back(matrix[SIZE * s + rc]);
    }

    std::sort(rows.begin(), rows.end());
    std::sort(cols.begin(), cols.end());

    if ((rows[0] == -1) || (cols[0] == -1))
      return 0;

    for (int i = 0; i < SIZE - 1; i++)
    {
      if ((rows[i] == rows[i + 1]) || (cols[i] == cols[i + 1]))
      {
        return 0;
      }
    }

    rows.clear();
    cols.clear();
  }

  return 1;
}

int main(int argc, char **argv)
{
  printf("in Oguzhan Ilter's futoshiki_hw3.cu \n");
  hipSetDevice(0);

  hipEvent_t start, stop;
  float elapsedTime;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // ------------------------------- Reading Data From txt Start -----------------------------------------
  std::string filename(argv[1]);
  std::ifstream file(filename.c_str());
  std::ifstream scout(filename.c_str());

  int no_grids;
  file >> no_grids;

  int dummy;
  scout >> dummy;

  int **grids = new int *[no_grids];
  int **constraints = new int *[no_grids];

  for (int i = 0; i < no_grids; i++)
  {
    grids[i] = new int[SIZE * SIZE];
  }

  int elem0, elem1, elem2, elem3, elem4;
  int pre_cursor = 0;
  int cursor = 0;
  int csize = 0;

  std::string file_line;
  std::string scout_line;

  int *constraint_sizes = new int[no_grids];
  int maxNumberofConstraint = 0;
  std::getline(scout, scout_line); // These are for spare lines
  std::getline(scout, scout_line);
  for (int i = 0; i < INPUTSIZE; i++)
  {
    std::getline(scout, scout_line);
    if (scout_line == "-------")
    {
      csize = i - pre_cursor - 5;
      constraint_sizes[cursor] = csize;
      if (csize > maxNumberofConstraint)
      {
        maxNumberofConstraint = csize;
      }
      cursor++;
      pre_cursor = i + 1;
    }
  }

  std::ofstream solved("solved.txt");

  solved << no_grids << std::endl;
  solved << "-------" << std::endl;

  for (int i = 0; i < no_grids; i++)
  {
    constraints[i] = new int[constraint_sizes[i] * 4];
  }

  std::getline(file, file_line);
  for (int i = 0; i < no_grids; i++)
  {
    std::getline(file, file_line);
    for (int j = 0; j < SIZE; j++)
    {
      std::getline(file, file_line);
      // std::cout << "i: " << i << " file_line: " << file_line << std::endl;
      std::istringstream iss(file_line);
      iss >> elem0 >> elem1 >> elem2 >> elem3 >> elem4;
      grids[i][SIZE * j + 0] = elem0;
      grids[i][SIZE * j + 1] = elem1;
      grids[i][SIZE * j + 2] = elem2;
      grids[i][SIZE * j + 3] = elem3;
      grids[i][SIZE * j + 4] = elem4;
    }
    for (int c = 0; c < constraint_sizes[i]; c++)
    {
      std::getline(file, file_line);
      // std::cout << "i: " << i << "c line: " << file_line << std::endl;
      std::istringstream iss(file_line);
      iss >> elem0 >> elem1 >> elem2 >> elem3;
      constraints[i][4 * c] = elem0;
      constraints[i][4 * c + 1] = elem1;
      constraints[i][4 * c + 2] = elem2;
      constraints[i][4 * c + 3] = elem3;
    }
  }

  long int *constraintsSums = new long int[no_grids + 1];
  constraintsSums[0] = 0;
  for (int i = 1; i < no_grids + 1; i++)
  {
    constraintsSums[i] += constraintsSums[i - 1] + constraint_sizes[i - 1];
  }

  int *constraintsArray = new int[4 * constraintsSums[no_grids]];
  int *gridsArray = new int[SIZE * SIZE * no_grids];

  for (int y = 0, k = 0, j = 0; y < no_grids; y++)
  {
    for (int z = 0; z < constraint_sizes[y]; z++)
    {
      constraintsArray[k++] = constraints[y][4 * z];
      constraintsArray[k++] = constraints[y][4 * z + 1];
      constraintsArray[k++] = constraints[y][4 * z + 2];
      constraintsArray[k++] = constraints[y][4 * z + 3];
    }

    for (int i = 0; i < SIZE * SIZE; i++)
    {
      gridsArray[j++] = grids[y][i];
    }
  }

  // ------------------------------- Reading Data From txt Done -----------------------------------------

  // ------------------------------- GPU Memcpy and Kernel Call -----------------------------------------

  int *matrix_in, *constraints_in, *numberOfInput;
  long int *constraintSums_in;
  size_t Smatrix_in = SIZE * SIZE * no_grids * sizeof(int);
  size_t Sconstraints_in = 4 * constraintsSums[no_grids] * sizeof(int);
  size_t SconstraintSums_in = (no_grids + 1) * sizeof(long int);

  hipEventRecord(start, 0);

  checkCuda(hipMalloc((void **)&matrix_in, Smatrix_in));
  checkCuda(hipMalloc((void **)&constraints_in, Sconstraints_in));
  checkCuda(hipMalloc((void **)&constraintSums_in, SconstraintSums_in));
  checkCuda(hipMalloc((void **)&numberOfInput, sizeof(int)));

  checkCuda(hipMemcpy(matrix_in, gridsArray, Smatrix_in, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(constraints_in, constraintsArray, Sconstraints_in, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(constraintSums_in, constraintsSums, SconstraintSums_in, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(numberOfInput, &no_grids, sizeof(int), hipMemcpyHostToDevice));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" Host2Device %lf sec ", elapsedTime / 1000);

  hipEventRecord(start, 0);
  solver<<<BLOCKSIZE, THREADPERBLOCK>>>(matrix_in, constraints_in, constraintSums_in, numberOfInput);

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" Kernel %lf sec ", elapsedTime / 1000);
  hipDeviceSynchronize();
  checkCuda(hipPeekAtLastError());

  hipEventRecord(start, 0);
  checkCuda(hipMemcpy(gridsArray, matrix_in, Smatrix_in, hipMemcpyDeviceToHost));

  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsedTime, start, stop);
  printf(" Device2Host %lf sec ", elapsedTime / 1000);
  // ------------------------------- **************************************************** -----------------------------------------

  // ------------------------------- Correctness Check and Solved Matrices txt Write Start-----------------------------------------
  int *matrix = new int[SIZE * SIZE];
  bool isAllCorrect = true;
  for (int index = 0; index < no_grids; index++)
  {
    for (int j = 0; j < 5; j++)
    {

      matrix[5 * j] = gridsArray[25 * index + 5 * j];
      matrix[5 * j + 1] = gridsArray[25 * index + 5 * j + 1];
      matrix[5 * j + 2] = gridsArray[25 * index + 5 * j + 2];
      matrix[5 * j + 3] = gridsArray[25 * index + 5 * j + 3];
      matrix[5 * j + 4] = gridsArray[25 * index + 5 * j + 4];

      solved << matrix[5 * j] << " " << matrix[5 * j + 1] << " " << matrix[5 * j + 2]
             << " " << matrix[5 * j + 3] << " " << matrix[5 * j + 4] << std::endl;
    }

    solved << "-------" << std::endl;

    if (!isSolved(matrix, constraints[index], constraint_sizes[index]))
    {
      printf("Puzzle %i is WRONG!!! \n", index);
      isAllCorrect = false;
    }
  }

  if (isAllCorrect)
  {
    printf(" \n ***ALL DONE !!! ALL CORRECT*** \n");
    printf("Output file name is solved.txt \n");
  }
  // ------------------------------- Correctness Check and Solved Matrices txt Write Done-----------------------------------------

  checkCuda(hipFree(matrix_in));
  checkCuda(hipFree(constraints_in));
  checkCuda(hipFree(constraintSums_in));

  return 0;
}